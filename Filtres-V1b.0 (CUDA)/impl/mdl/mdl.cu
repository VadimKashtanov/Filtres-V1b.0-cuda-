#include "mdl.cuh"

PAS_OPTIMISER()
Mdl_t * cree_mdl(
	uint BLOQUES, uint BLOQUE_DIM, uint * BLOQUE,
	ema_int * arr_ema_int,
	uint PENSEE_DIM, uint * PENSEE)
{
	ASSERT(BLOQUE[BLOQUE_DIM-1] == 1);
	//ASSERT(PENSEE[PENSEE_DIM-1] == 3);
	Mdl_t * mdl = (Mdl_t*)malloc(sizeof(Mdl_t));

	mdl->BLOQUES = BLOQUES;
	mdl->C_BLOQUE = BLOQUE_DIM;
	mdl->BLOQUE = cpyuint(BLOQUE, BLOQUE_DIM);

	mdl->arr_ema_int = (ema_int*)malloc(sizeof(ema_int) * BLOQUE[0] * BLOQUES);
	memcpy(mdl->arr_ema_int, arr_ema_int, sizeof(ema_int) * BLOQUE[0] * BLOQUES);

	mdl->C_PENSEE = PENSEE_DIM;
	mdl->PENSEE = cpyuint(PENSEE, PENSEE_DIM);

	mdl->FILTRES = BLOQUE[0] * BLOQUES * N;
	mdl->POIDS = 0;
	mdl->VARS  = 0;
	mdl->LOCDS = 0;

	mdl->DEPART_POIDS = (uint*)malloc(sizeof(uint) * COUCHES);
	mdl->DEPART_VARS  = (uint*)malloc(sizeof(uint) * COUCHES);
	mdl->DEPART_LOCDS = (uint*)malloc(sizeof(uint) * COUCHES);

	//	Instruction: Bloque
	FOR(0, i, mdl->C_BLOQUE) {
		ASSERT(mdl->BLOQUE[i]*mdl->BLOQUES <= MAX_Y);
		mdl->DEPART_VARS [i] = mdl->VARS ;
		mdl->DEPART_POIDS[i] = mdl->POIDS;
		mdl->DEPART_LOCDS[i] = mdl->LOCDS;
		//
		mdl->VARS  +=                 BLOQUE[i]                * BLOQUES;
		mdl->POIDS += (i == 0 ? 0 : ((BLOQUE[i-1]+1)*BLOQUE[i])) * BLOQUES;
		mdl->LOCDS += (        i == 0 ? 0 : BLOQUE[i]        ) * BLOQUES;
	};

	//	Instruction: Pensee
	FOR(0, i, mdl->C_PENSEE) {
		ASSERT(mdl->PENSEE[i] <= MAX_Y);

		mdl->DEPART_VARS [mdl->C_BLOQUE+i] = mdl->VARS ;
		mdl->DEPART_POIDS[mdl->C_BLOQUE+i] = mdl->POIDS;
		mdl->DEPART_LOCDS[mdl->C_BLOQUE+i] = mdl->LOCDS;
		//
		//printf("vars=%i +%i\n", mdl->VARS, PENSEE[i]);
		mdl->VARS  += PENSEE[i];
		mdl->POIDS += (i == 0 ? ((BLOQUES+1)*PENSEE[0]) : ((PENSEE[i-1]+1)*PENSEE[i]));
		mdl->LOCDS += PENSEE[i];
	};

	//	======= Allocation ========
	mdl->f    = (float*)malloc(sizeof(float) * mdl->FILTRES);
	mdl->p    = (float*)malloc(sizeof(float) * mdl->POIDS);
	mdl->y    = (float*)malloc(sizeof(float) * mdl->VARS);
	mdl->locd = (float*)malloc(sizeof(float) * mdl->LOCDS);

	CONTROLE_CUDA(hipMalloc((void**)&mdl->f_d,    sizeof(float) * mdl->FILTRES));
	CONTROLE_CUDA(hipMalloc((void**)&mdl->p_d,    sizeof(float) * mdl->POIDS));
	//CONTROLE_CUDA(hipMalloc((void**)&mdl->y_d,    sizeof(float) * mdl->VARS));
	//CONTROLE_CUDA(hipMalloc((void**)&mdl->locd_d, sizeof(float) * mdl->LOCDS));

	CONTROLE_CUDA(hipMalloc((void**)&mdl->dp_d, sizeof(float) * mdl->POIDS);)
	CONTROLE_CUDA(hipMalloc((void**)&mdl->dy_d, sizeof(float) * mdl->VARS));

	mdl->dif_f = (float*)malloc(sizeof(float) * BLOQUE[0] * BLOQUES * (N-1));
	CONTROLE_CUDA(hipMalloc((void**)&mdl->dif_f_d, sizeof(float) * BLOQUE[0] * BLOQUES * (N-1)));

	FOR(0, i, mdl->FILTRES) mdl->f[i] = rnd();
	FOR(0, i, mdl->POIDS) mdl->p[i] = 2*rnd()-1;
	FOR(0, i, mdl->FILTRES / N) normer(mdl->f + i*N, N);

	//	Qlq uint pour cuda
	CONTROLE_CUDA(hipMalloc((void**)&mdl->DEPART_POIDS__d, sizeof(float)*(mdl->C_BLOQUE+mdl->C_PENSEE)));
	CONTROLE_CUDA(hipMalloc((void**)&mdl->PENSEE__d, sizeof(float)*mdl->C_PENSEE));
	CONTROLE_CUDA(hipMalloc((void**)&mdl->BLOQUE__d, sizeof(float)*mdl->C_BLOQUE));
	CONTROLE_CUDA(hipMalloc((void**)&mdl->arr_ema_int__d, sizeof(ema_int)*mdl->BLOQUES*mdl->BLOQUE[0]));
	//
	CONTROLE_CUDA(hipMemcpy(mdl->DEPART_POIDS__d, mdl->DEPART_POIDS, sizeof(float)*(mdl->C_BLOQUE+mdl->C_PENSEE), hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(mdl->PENSEE__d, 	  mdl->PENSEE,       sizeof(float)*mdl->C_PENSEE, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(mdl->BLOQUE__d,       mdl->BLOQUE,       sizeof(float)*mdl->C_BLOQUE, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(mdl->arr_ema_int__d,  mdl->arr_ema_int,  sizeof(ema_int)*mdl->BLOQUES*mdl->BLOQUE[0], hipMemcpyHostToDevice));
	//

	prep_mdl(mdl);

	return mdl;
};

void mdl_liberer(Mdl_t * mdl) {
	free(mdl->BLOQUE);
	free(mdl->PENSEE);
	free(mdl->DEPART_POIDS);
	free(mdl->DEPART_VARS);
	free(mdl->DEPART_LOCDS);
	//
	free(mdl->arr_ema_int);
	//
	free(mdl->dif_f);
	//
	free(mdl->f);
	free(mdl->p);
	free(mdl->y);
	free(mdl->locd);
	//
	CONTROLE_CUDA(hipFree(mdl->f_d));
	CONTROLE_CUDA(hipFree(mdl->p_d));
	//CONTROLE_CUDA(hipFree(mdl->y_d));
	//CONTROLE_CUDA(hipFree(mdl->locd_d));
	CONTROLE_CUDA(hipFree(mdl->dp_d));
	CONTROLE_CUDA(hipFree(mdl->dy_d));
	CONTROLE_CUDA(hipFree(mdl->dif_f_d));
	//
	CONTROLE_CUDA(hipFree(mdl->DEPART_POIDS__d));
	CONTROLE_CUDA(hipFree(mdl->PENSEE__d));
	CONTROLE_CUDA(hipFree(mdl->BLOQUE__d));
	CONTROLE_CUDA(hipFree(mdl->arr_ema_int__d));
};

void prep_mdl(Mdl_t * mdl) {
	memset(mdl->y, 0, sizeof(float) * mdl->VARS);
	memset(mdl->locd, 0, sizeof(float) * mdl->LOCDS);

	CONTROLE_CUDA(hipMemcpy(mdl->p_d, mdl->p, sizeof(float)*mdl->POIDS, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(mdl->f_d, mdl->f, sizeof(float)*mdl->FILTRES, hipMemcpyHostToDevice));

	//CONTROLE_CUDA(hipMemset(mdl->y_d,     0, sizeof(float) * mdl->VARS));
	//CONTROLE_CUDA(hipMemset(mdl->locd_d,  0, sizeof(float) * mdl->LOCDS));
	CONTROLE_CUDA(hipMemset(mdl->dp_d,    0, sizeof(float) * mdl->POIDS));
	CONTROLE_CUDA(hipMemset(mdl->dy_d,    0, sizeof(float) * mdl->VARS));
	
	uint i;
//#pragma omp parallel for private(i)
	for (i=0; i < mdl->BLOQUE[0] * mdl->BLOQUES; i++) {
		FOR(0, j, N-1) {
			mdl->dif_f[i*(N-1) + j] = mdl->f[i*N+j+1]-mdl->f[i*N+j];
		}
	}
	hipMemcpy(mdl->dif_f_d, mdl->dif_f, sizeof(float)*mdl->BLOQUE[0] * mdl->BLOQUES * (N-1), hipMemcpyHostToDevice);
};

void gpu_vers_cpu(Mdl_t * mdl) {
	CONTROLE_CUDA(hipMemcpy(mdl->p, mdl->p_d, sizeof(float)*mdl->POIDS, hipMemcpyDeviceToHost));
	CONTROLE_CUDA(hipMemcpy(mdl->f, mdl->f_d, sizeof(float)*mdl->FILTRES, hipMemcpyDeviceToHost));
};

void reinit_mdl(Mdl_t * mdl) {
	prep_mdl(mdl);
};