#include "hip/hip_runtime.h"
#include "mdl.cuh"

static float filtre(float * x, float * dif_x, float * f, float * dif_f) {
	float s = 0, d = 0;
	FOR(0, i, N-1) {
		s += sqrtf(1 + fabs(  x[i]   -   f[i]  ));
		d += powf((1 + fabs(dif_x[i] - dif_f[i])), 2);
	};
	s += sqrtf(1 + fabs(x[N-1] - f[N-1]));

	s = s/8-1;
	d = d/7-1;

	return 2*expf(-s*s -d*d)-1;
};

static float perceptron(float * x, float * p, uint _N) {
	float s = p[_N-1+1];
	FOR(0, i, _N) s += x[i]*p[i];
	return ACTIV(s);
};

float f(Mdl_t * mdl, uint t) {
	//	Filtres
	FOR(0, i, mdl->BLOQUE[0] * mdl->BLOQUES) {
		mdl->y[i] = filtre(
			    normalisee + mdl->arr_ema_int[i].ligne*PRIXS*N_FLTR + t*N_FLTR,
			dif_normalisee + mdl->arr_ema_int[i].ligne*PRIXS*N_FLTR + t*N_FLTR,
			mdl->f + i*N,
			mdl->dif_f + i*(N-1)
		);
	}
	
	//	Bloques
	FOR(1, c, mdl->C_BLOQUE) {
		FOR(0, b, mdl->BLOQUES) {
			FOR(0, y, mdl->BLOQUE[c]) {
				mdl->y[mdl->DEPART_VARS[c] + b*mdl->BLOQUE[c] + y] = perceptron(
					mdl->y + mdl->DEPART_VARS[c-1] + b*mdl->BLOQUE[c-1],
					mdl->p +  mdl->DEPART_POIDS[c] + (b*mdl->BLOQUE[c] + y)*(mdl->BLOQUE[c-1]+1),
					mdl->BLOQUE[c-1]
				);
				//printf("%f ", mdl->y[mdl->DEPART_VARS[c] + b*mdl->BLOQUE[c] + y]);
			};
		};
		//printf("\n");
	};

	//	Pensee
	FOR(0, c, mdl->C_PENSEE) {
		FOR(0, y, mdl->PENSEE[c]) {
			uint qt_vars_couche_pred = (c == 0 ? mdl->BLOQUES*1 : mdl->PENSEE[c-1]);
			mdl->y[mdl->DEPART_VARS[mdl->C_BLOQUE+c] + y] = perceptron(
				mdl->y + mdl->DEPART_VARS[mdl->C_BLOQUE+c-1],
				mdl->p + mdl->DEPART_POIDS[mdl->C_BLOQUE+c] + y*(qt_vars_couche_pred+1),
				(c == 0 ? mdl->BLOQUES : mdl->PENSEE[c-1])
			);
		};
	};
	return mdl->y[mdl->VARS-1];
};

void  cpu_mdl_f_t0t1(Mdl_t * mdl, float * res, uint t0, uint t1) {
	uint t;
//#pragma omp parallel for private(t)  //(Incoherances et inexactitudes !)
	for (t=t0; t < t1; t++) {
		res[t-t0] = f(mdl, t);
	}
};
