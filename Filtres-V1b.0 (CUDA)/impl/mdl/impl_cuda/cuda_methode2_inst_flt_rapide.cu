#include "hip/hip_runtime.h"
#include "mdl.cuh"

#define MULTIPLE 1 //4 avant

static __global__ void cart_graphique_kerd__FILTRES(
	uint * ptr_vers_pos__d,
	uint LIGNE,
	//
	float * y_d, uint C_MAX,
	ema_int * arr_ema_int__d,
	float * normalisee__d, float * dif_normalisee__d,
	float * f_d, float * dif_f_d,
	uint TAILLE_BASSIN, uint T,
	uint t0)
{
	// <<< (x, t) >>>
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint t = /*threadIdx.y + */blockIdx.y/* * blockDim.y*/;

	__shared__ float     flt__x[N_FLTR];
	__shared__ float dif_flt__x[N_FLTR];
	//
	uint reel_t = t0 + t;
	//
	flt__x[x] = normalisee__d[LIGNE*PRIXS*N_FLTR + reel_t*N_FLTR + threadIdx.x];
	dif_flt__x[x] = dif_normalisee__d[LIGNE*PRIXS*N_FLTR + reel_t*N_FLTR + threadIdx.x];
	//
	if (t < T && x < TAILLE_BASSIN) {
		//uint reel_t = t0 + t;
		//
		float s = 0, d = 0;
		FOR(0, i, N-1) {
			s += sqrtf((1.f + fabs(
				flt__x[i] - f_d[x*N+i]
			)));
			d += powf((1.f + fabs(
				dif_flt__x[i] - dif_f_d[x*(N-1)+i]
			)), 2);
		};
		s += sqrtf(1.f + fabs(
			flt__x[N-1] - f_d[x*N + (N-1)]
		));
		s = s/8-1;
		d = d/7-1;
		//
		y_d[reel_t*C_MAX + ptr_vers_pos__d[x]] = 2*expf(-s*s -d*d)-1;
	}
};

static __global__ void cart_graphique_kerd__perceptron_BLOQUE(
	uint * DEPART_POIDS__d, uint c,
	uint BLOQUES, uint X, uint Y,
	float * x_d, float * y_d, uint C_MAX,
	float * p_d,
	uint TS)
{
	//	<<<(Bloque, t, y)>>>
	uint Bx = threadIdx.x + blockIdx.x * blockDim.x;
	uint Ty = threadIdx.y + blockIdx.y * blockDim.y;
	uint Yz = threadIdx.z + blockIdx.z * blockDim.z;

	if (((Bx < BLOQUES) && (Ty < TS)) && (Yz < Y)) {
		float s = p_d[DEPART_POIDS__d[c] + Bx*Y*(X+1) + Yz*(X+1) + (X-1)+1];
		FOR(0, i, X)
			s += x_d[Ty*C_MAX + Bx*X + i] * p_d[DEPART_POIDS__d[c] + Bx*Y*(X+1) + Yz*(X+1) + i];
		y_d[Ty*C_MAX + Bx*Y + Yz] = tanh(s);
	}
}

static __global__ void cart_graphique_kerd__perceptron_PENSEE(
	uint X, uint Y,
	float * x_d, float * y_d, uint C_MAX,
	float * p_d,
	uint TS)
{
	// <<< (y, t) >>>
	uint Yx = threadIdx.x + blockIdx.x * blockDim.x;
	uint Ty = threadIdx.y + blockIdx.y * blockDim.y;

	if (Ty < TS && Yx < Y) {
		float s = p_d[Yx*(X+1) + (X-1)+1];
		FOR(0, i, X) s += x_d[Ty*C_MAX + i] * p_d[Yx*(X+1) + i];
		y_d[Ty*C_MAX + Yx] = tanh(s);
		//printf("%f ", tanh(s));
	}
}

static __global__ void enregistrer_les_resultats(
	float * res_d, float * y,
	uint C_MAX, uint TS)
{
	uint Tx = threadIdx.x + blockIdx.x * blockDim.x;

	if (Tx < TS) {
		res_d[Tx] = y[Tx*C_MAX + 0];
	};
};

void cuda_mdt2_mdl_f_t0t1(Mdl_t * mdl, float * res, uint t0, uint t1) {
	float * res_d;
	CONTROLE_CUDA(hipMalloc((void**)&res_d, sizeof(float)*(t1-t0)));
	CONTROLE_CUDA(hipMemset(res_d, 0, sizeof(float)*(t1-t0)));
	
	//
	uint C_MAX = mdl->BLOQUE[0] * mdl->BLOQUES;
	FOR(1, i, mdl->C_BLOQUE) if (mdl->BLOQUE[i]*mdl->BLOQUES > C_MAX) C_MAX = mdl->BLOQUE[i]*mdl->BLOQUES;
	FOR(0, i, mdl->C_PENSEE) if (mdl->PENSEE[i] > C_MAX) C_MAX = mdl->PENSEE[i];

	float * r0__d;
	float * r1__d;
	CONTROLE_CUDA(hipMalloc((void**)&r0__d, sizeof(float) * PRIXS * C_MAX));
	CONTROLE_CUDA(hipMalloc((void**)&r1__d, sizeof(float) * PRIXS * C_MAX));

	//	--- Filtres ---
	uint FS = mdl->BLOQUES * mdl->BLOQUE[0];
	uint TS = t1 - t0;
	//
	//	On fait `EMA_INTS` bassins qui utiliseront __shared__ pour calc les filtres
	//
	uint TAILLES_BASSINS[EMA_INTS]     = {0};
	uint         BASSINS[EMA_INTS][FS] = {0};
	uint *    BASSINS__d[EMA_INTS]     = {0};
	//
	FOR(0, i, FS) {
		uint ligne           = mdl->arr_ema_int[i].ligne;
		uint pos_dans_bassin =    TAILLES_BASSINS[ligne];
		BASSINS[  ligne ][ pos_dans_bassin ] = i;
		  TAILLES_BASSINS[       ligne     ] ++ ;
	}
	//
	FOR(0, i, EMA_INTS) {
		CONTROLE_CUDA(hipMalloc((void**)&BASSINS__d[i], sizeof(uint)*FS));
		CONTROLE_CUDA(hipMemcpy(BASSINS__d[i], BASSINS[i], sizeof(uint)*FS, hipMemcpyHostToDevice));
	}
	//
	//	MULTIPLE*N_FLTR devrait etre dans les 16, 32, 64
	//	chaque MULTIPLE-ieme Id.x CHARGERA 1 valeure %N_FLT dans la __shared__
	FOR(0, i, EMA_INTS)
	{
		if (!(TAILLES_BASSINS[i] == 0))
		{
			dim3 grille(KER_DIV(TAILLES_BASSINS[i], MULTIPLE*N_FLTR), KER_DIV(TS, 1));
			dim3 bloque(      MULTIPLE*N_FLTR,                       1       );
			cart_graphique_kerd__FILTRES<<<grille, bloque>>>(
				BASSINS__d[i],
				i,
				//
				r0__d, C_MAX,
				mdl->arr_ema_int__d,
				normalisee__d, dif_normalisee__d,
				mdl->f_d, mdl->dif_f_d,
				TAILLES_BASSINS[i], TS,
				t0
			);
		}
	}
	ATTENDRE_KER_CUDA();

	FOR(0, i, EMA_INTS) CONTROLE_CUDA(hipFree(BASSINS__d[i]));

	//	--- Perceptrons des Bloques ---
	FOR(1, c, mdl->C_BLOQUE) {
		cart_graphique_kerd__perceptron_BLOQUE<<<dim3(KER_DIV(mdl->BLOQUES, 8), KER_DIV(TS, 8), KER_DIV(mdl->BLOQUE[c], 8)), dim3(8,8,8)>>>(
			mdl->DEPART_POIDS__d, c,
			mdl->BLOQUES, mdl->BLOQUE[c-1], mdl->BLOQUE[c],
			(c%2==0 ? r1__d : r0__d), (c%2==0 ? r0__d : r1__d), C_MAX,
			mdl->p_d,
			TS
		);
		ATTENDRE_KER_CUDA();
	}

	//	--- Perceptron de la Pensee ---
	FOR(0, c, mdl->C_PENSEE) {
		cart_graphique_kerd__perceptron_PENSEE<<<dim3(KER_DIV(mdl->PENSEE[c], 32), KER_DIV(TS, 32)),dim3(32,32)>>>(
			(c==0 ? mdl->BLOQUES : mdl->PENSEE[c-1]), mdl->PENSEE[c],
			((mdl->C_BLOQUE+c)%2==0 ? r1__d : r0__d), ((mdl->C_BLOQUE+c)%2==0 ? r0__d : r1__d), C_MAX,
			mdl->p_d + mdl->DEPART_POIDS[mdl->C_BLOQUE + c],
			TS
		);
		ATTENDRE_KER_CUDA();
	}

	//	--- Resultat ---
	enregistrer_les_resultats<<<dim3(KER_DIV(TS,1024)), dim3(1024)>>>(
		res_d, ((mdl->C_BLOQUE+mdl->C_PENSEE)%2==0 ? r1__d : r0__d),
		C_MAX, TS);
	ATTENDRE_KER_CUDA();

	//	Sortie
	CONTROLE_CUDA(hipMemcpy(
		res,
		res_d,
		sizeof(float)*(t1-t0),
		hipMemcpyDeviceToHost
	));
	CONTROLE_CUDA(hipFree(res_d));
	CONTROLE_CUDA(hipFree(r0__d));
	CONTROLE_CUDA(hipFree(r1__d));
};