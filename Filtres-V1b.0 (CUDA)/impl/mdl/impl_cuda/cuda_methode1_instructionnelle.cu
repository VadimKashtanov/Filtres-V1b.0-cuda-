#include "hip/hip_runtime.h"
#include "mdl.cuh"

static __global__ void cart_graphique_kerd__FILTRES(
	float * y_d, uint C_MAX,
	ema_int * arr_ema_int__d,
	float * normalisee__d, float * dif_normalisee__d,
	float * f_d, float * dif_f_d,
	uint FS, uint TS,
	uint t0)
{
	// <<< (filtre, t) >>>
	uint Fx = threadIdx.x + blockIdx.x * blockDim.x;
	uint Ty = threadIdx.y + blockIdx.y * blockDim.y;

	if (Ty < TS && Fx < FS) {
		uint t = t0 + Ty;
		//
		float s = 0, d = 0;
		uint ligne = arr_ema_int__d[Fx].ligne;
		FOR(0, i, N-1) {
			s += sqrtf((1.f + fabs(
				    normalisee__d[ligne*PRIXS*N_FLTR + t*N_FLTR + i] - /*const_mem[Fx*N+i]*/f_d[Fx*N+i]
			)));
			d += powf((1.f + fabs(
				dif_normalisee__d[ligne*PRIXS*N_FLTR + t*N_FLTR + i] - /*const_mem[FS*N + Fx*(N-1)+i]*/dif_f_d[Fx*(N-1)+i]
			)), 2);
		};
		s += sqrtf(1.f + fabs(
			normalisee__d[ligne*PRIXS*N_FLTR + t*N_FLTR + (N-1)] - /*const_mem[Fx*N + (N-1)]*/f_d[Fx*N + (N-1)]
		));
		s = s/8-1;
		d = d/7-1;
		//
		y_d[Ty*C_MAX + Fx] = 2*expf(-s*s -d*d)-1;
	}
};

static __global__ void cart_graphique_kerd__perceptron_BLOQUE(
	uint * DEPART_POIDS__d, uint c,
	uint BLOQUES, uint X, uint Y,
	float * x_d, float * y_d, uint C_MAX,
	float * p_d,
	uint TS)
{
	//	<<<(Bloque, t, y)>>>
	uint Bx = threadIdx.x + blockIdx.x * blockDim.x;
	uint Ty = threadIdx.y + blockIdx.y * blockDim.y;
	uint Yz = threadIdx.z + blockIdx.z * blockDim.z;

	if (((Bx < BLOQUES) && (Ty < TS)) && (Yz < Y)) {
		float s = p_d[DEPART_POIDS__d[c] + Bx*Y*(X+1) + Yz*(X+1) + (X-1)+1];
		FOR(0, i, X)
			s += x_d[Ty*C_MAX + Bx*X + i] * p_d[DEPART_POIDS__d[c] + Bx*Y*(X+1) + Yz*(X+1) + i];
		y_d[Ty*C_MAX + Bx*Y + Yz] = tanh(s);
	}
}

static __global__ void cart_graphique_kerd__perceptron_PENSEE(
	uint X, uint Y,
	float * x_d, float * y_d, uint C_MAX,
	float * p_d,
	uint TS)
{
	// <<< (y, t) >>>
	uint Yx = threadIdx.x + blockIdx.x * blockDim.x;
	uint Ty = threadIdx.y + blockIdx.y * blockDim.y;

	if (Ty < TS && Yx < Y) {
		float s = p_d[Yx*(X+1) + (X-1)+1];
		FOR(0, i, X) s += x_d[Ty*C_MAX + i] * p_d[Yx*(X+1) + i];
		y_d[Ty*C_MAX + Yx] = tanh(s);
		//printf("%f ", tanh(s));
	}
}

static __global__ void enregistrer_les_resultats(
	float * res_d, float * y,
	uint C_MAX, uint TS)
{
	uint Tx = threadIdx.x + blockIdx.x * blockDim.x;

	if (Tx < TS) {
		res_d[Tx] = y[Tx*C_MAX + 0];
	};
};

void cuda_mdt1_mdl_f_t0t1(Mdl_t * mdl, float * res, uint t0, uint t1) {
	float * res_d;
	CONTROLE_CUDA(hipMalloc((void**)&res_d, sizeof(float)*(t1-t0)));
	CONTROLE_CUDA(hipMemset(res_d, 0, sizeof(float)*(t1-t0)));
	
	//
	uint C_MAX = mdl->BLOQUE[0] * mdl->BLOQUES;
	FOR(1, i, mdl->C_BLOQUE) if (mdl->BLOQUE[i]*mdl->BLOQUES > C_MAX) C_MAX = mdl->BLOQUE[i]*mdl->BLOQUES;
	FOR(0, i, mdl->C_PENSEE) if (mdl->PENSEE[i] > C_MAX) C_MAX = mdl->PENSEE[i];

	float * r0__d;
	float * r1__d;
	CONTROLE_CUDA(hipMalloc((void**)&r0__d, sizeof(float) * PRIXS * C_MAX));
	CONTROLE_CUDA(hipMalloc((void**)&r1__d, sizeof(float) * PRIXS * C_MAX));

	//CONTROLE_CUDA(hipMemset(r0__d, 0, sizeof(float) * PRIXS * C_MAX));
	//CONTROLE_CUDA(hipMemset(r1__d, 0, sizeof(float) * PRIXS * C_MAX));

	//	--- Filtres ---
	uint FS = mdl->BLOQUES * mdl->BLOQUE[0];
	uint TS = t1 - t0;
	cart_graphique_kerd__FILTRES<<<dim3(KER_DIV(FS, 16), KER_DIV(TS, 32)),dim3(16,32)>>>(
		r0__d, C_MAX,
		mdl->arr_ema_int__d,
		normalisee__d, dif_normalisee__d,
		mdl->f_d, mdl->dif_f_d,
		FS, TS,
		t0
	);
	ATTENDRE_KER_CUDA();

	//	--- Perceptrons des Bloques ---
	FOR(1, c, mdl->C_BLOQUE) {
		cart_graphique_kerd__perceptron_BLOQUE<<<dim3(KER_DIV(mdl->BLOQUES, 8), KER_DIV(TS, 8), KER_DIV(mdl->BLOQUE[c], 8)), dim3(8,8,8)>>>(
			mdl->DEPART_POIDS__d, c,
			mdl->BLOQUES, mdl->BLOQUE[c-1], mdl->BLOQUE[c],
			(c%2==0 ? r1__d : r0__d), (c%2==0 ? r0__d : r1__d), C_MAX,
			mdl->p_d,// + mdl->DEPART_POIDS[c],
			TS
		);
		ATTENDRE_KER_CUDA();
		//printf("\n");
	}

	//	--- Perceptron de la Pensee ---
	FOR(0, c, mdl->C_PENSEE) {
		cart_graphique_kerd__perceptron_PENSEE<<<dim3(KER_DIV(mdl->PENSEE[c], 16), KER_DIV(TS, 64)),dim3(16,64)>>>(
			(c==0 ? mdl->BLOQUES : mdl->PENSEE[c-1]), mdl->PENSEE[c],
			((mdl->C_BLOQUE+c)%2==0 ? r1__d : r0__d), ((mdl->C_BLOQUE+c)%2==0 ? r0__d : r1__d), C_MAX,
			mdl->p_d + mdl->DEPART_POIDS[mdl->C_BLOQUE + c],
			TS
		);
		ATTENDRE_KER_CUDA();
	}

	//	--- Resultat ---
	enregistrer_les_resultats<<<dim3(KER_DIV(TS,1024)), dim3(1024)>>>(
		res_d, ((mdl->C_BLOQUE+mdl->C_PENSEE)%2==0 ? r1__d : r0__d),
		C_MAX, TS);
	ATTENDRE_KER_CUDA();

	//	Sortie
	CONTROLE_CUDA(hipMemcpy(
		res,
		res_d,
		sizeof(float)*(t1-t0),
		hipMemcpyDeviceToHost
	));
	CONTROLE_CUDA(hipFree(res_d));
	CONTROLE_CUDA(hipFree(r0__d));
	CONTROLE_CUDA(hipFree(r1__d));
};