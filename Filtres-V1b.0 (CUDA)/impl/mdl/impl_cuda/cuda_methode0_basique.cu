#include "hip/hip_runtime.h"
#include "mdl.cuh"

/*	Filtres : Memoire constante
	Poids   : Memoire constante
*/

__device__ static float perceptron(float * x, float * p, uint _N) {
	float s = p[_N-1+1];
	FOR(0, i, _N) {
		s += x[i]*p[i];
	}
	return ACTIV(s);
};

__global__ void kerd_mdl(
	uint  BLOQUES,	
	uint C_BLOQUE,
	uint * BLOQUE__d,
	ema_int * arr_ema_int__d,
	uint C_PENSEE,
	uint * PENSEE__d,
	uint * DEPART_POIDS__d, uint * DEPART_VARS__d, uint * DEPART_LOCDS__d,
	float * f_d, float * p_d,
	float * dif_f_d,
	uint t0, uint t1, float * res_d,
	float * normalisee__d, float * dif_normalisee__d)
{
	uint t = t0 + (threadIdx.x + blockIdx.x * blockDim.x);
	//
	if (t < t1) {
		float r0[MAX_Y];
		float r1[MAX_Y];

		//	FILTRES
		float s,d;
		uint ligne;
		FOR(0, f, BLOQUES*BLOQUE__d[0]) {
			ligne = arr_ema_int__d[f].ligne;
			s = 0; d = 0;
			FOR(0, i, N-1) {
				s += sqrtf(1 + fabs(
					    normalisee__d[ligne*PRIXS*N_FLTR + t*N_FLTR + i] - f_d[f*N+i]
				));
				d += powf((1 + fabs(
					dif_normalisee__d[ligne*PRIXS*N_FLTR + t*N_FLTR + i] - dif_f_d[f*(N-1)+i]
				)), 2);
			};
			s += sqrtf(1 + fabs(
				normalisee__d[ligne*PRIXS*N_FLTR + t*N_FLTR + (N-1)] - f_d[f*N + (N-1)]
			));
			s = s/8-1;
			d = d/7-1;
			r0[f] = 2*expf(-s*s -d*d)-1;
		}
		
		//	Triangles des Bloques
		FOR(1, c, C_BLOQUE) {
			FOR(0, b, BLOQUES) {
				FOR(0, y, BLOQUE__d[c]) {
					r1[b*BLOQUE__d[c] + y] = perceptron(
						r0 + b*BLOQUE__d[c-1],
						p_d + DEPART_POIDS__d[c] + (b*BLOQUE__d[c] + y)*(BLOQUE__d[c-1]+1),
						BLOQUE__d[c-1]
					);
				};
			}
			FOR(0, y, BLOQUE__d[c] * BLOQUES) {
				r0[y] = r1[y];
			}
		};

		//	Pensee
		FOR(0, c, C_PENSEE) {
			FOR(0, y, PENSEE__d[c]) {
				uint qt_vars_couche_pred = (c == 0 ? BLOQUES*1 : PENSEE__d[c-1]);
				r1[y] = perceptron(
					r0,
					p_d + DEPART_POIDS__d[C_BLOQUE+c] + y*(qt_vars_couche_pred+1),
					(c == 0 ? BLOQUES : PENSEE__d[c-1])
				);
			};
			FOR(0, y, PENSEE__d[c]) r0[y] = r1[y];
		};
		res_d[t-t0] = r0[0];
	};
};

void cuda_mdt0_mdl_f_t0t1(Mdl_t * mdl, float * res, uint t0, uint t1) {
	float * res_d;
	CONTROLE_CUDA(hipMalloc((void**)&res_d, sizeof(float)*(t1-t0)));
	CONTROLE_CUDA(hipMemset(res_d, 0, sizeof(float)*(t1-t0)));

	//	--- Mdl_t ---
	kerd_mdl<<<dim3(KER_DIV((t1-t0), 128)), dim3(128)>>>(
		mdl->BLOQUES,	
		mdl->C_BLOQUE,
		mdl->BLOQUE__d,
		mdl->arr_ema_int__d,
		mdl->C_PENSEE,
		mdl->PENSEE__d,
		mdl->DEPART_POIDS__d, mdl->DEPART_VARS__d, mdl->DEPART_LOCDS__d,
		mdl->f_d,
		mdl->p_d,
		mdl->dif_f_d,
		t0, t1, res_d,
		normalisee__d, dif_normalisee__d
	);
	ATTENDRE_KER_CUDA();

	CONTROLE_CUDA(hipMemcpy(
		res,
		res_d,
		sizeof(float)*(t1-t0),
		hipMemcpyDeviceToHost
	));

	//
	CONTROLE_CUDA(hipFree(res_d));
};