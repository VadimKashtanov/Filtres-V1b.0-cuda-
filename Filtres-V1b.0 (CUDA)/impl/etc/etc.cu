#include "hip/hip_runtime.h"
#include "etc.cuh"

double secondes() {
	struct timespec now;
	timespec_get(&now, TIME_UTC);
	return 1000.0*(((int64_t) now.tv_sec) * 1000 + ((int64_t) now.tv_nsec) / 1000000);
};

const uint factoriel[13] = { 1, 1, 2, 6, 24, 120, 720, 5040, 40320, 
                                    362880, 3628800, 39916800, 479001600 };

float rnd() {
	return (float)(rand()%100000)/100000.0;	//rand()%100 pour avoire 1.0
};

float ___exp(float x)  // cubic spline approximation
{
    union { float f; int i; } reinterpreter;

    reinterpreter.i = (int)(12102203.0f*x) + 127*(1 << 23);
    int m = (reinterpreter.i >> 7) & 0xFFFF;  // copy mantissa
    // empirical values for small maximum relative error (8.34e-5):
    reinterpreter.i +=
         ((((((((1277*m) >> 14) + 14825)*m) >> 14) - 79749)*m) >> 11) - 626;
    return reinterpreter.f;
}

float ___gauss(float x) {return ___exp(-x*x);};
float ___d_gauss(float x) {return -2*x*___gauss(x);};

float ___logistique(float x) {return 1.0/(1.0+___exp(-x));};
float ___d_logistique(float x) {return ___logistique(x)*(1 - ___logistique(x));};

float ___tanh(float x) {return tanhf(x);};
float ___d_tanh(float x) {return 1 - powf(___tanh(x), 2);};


float signe(float x) {return (x>=0 ? 1:-1);};

void gnuplot(float * arr, uint len, char * titre) {
	char buff[200];
	//
	FILE * fp = fopen("gnuplot_dat.dat", "w");
	//
	for (uint i=0; i < len; i++) {
		snprintf(buff, 100, "%i ", i);
		fputs(buff, fp);
		//
		snprintf(buff, 100, "%f\n", arr[i]);
		fputs(buff, fp);
	}
	fclose(fp);
	//
	snprintf(
		buff,
		200,
		"gnuplot -p -e \"set title \'%s\'; plot 'gnuplot_dat.dat' w lp\"",
		titre);
	//
	assert(!system(buff));
	//
	assert(!system("rm gnuplot_dat.dat"));
};

uint* cpyuint(uint * arr, uint len) {
	uint * ret = (uint*)malloc(sizeof(uint) * len);
	memcpy(ret, arr, sizeof(uint) * len);
	return ret;
}

float* allouer_flotants(uint nb) {
	return (float*)malloc(sizeof(float) * nb);
}

uint u_max(uint * x, uint len) {
	uint _max=x[0];
	FOR(1,i,len) {
		if (x[i] > _max)
			_max = x[i];
	};
	return _max;
}

PAS_OPTIMISER()
void normer(float * arr, uint n) {
	float max=arr[0], min=arr[0];
	FOR(1, i, n) {
		if (arr[i] > max) max = arr[i];
		if (arr[i] < min) min = arr[i];
		//printf("%f ", arr[i]);
	}
	//printf("\n");
	FOR(0, i , n) {
	//	printf("%f %f %f\n", arr[i], min, max);
		arr[i] = (arr[i]-min)/(max-min);
		assert(arr[i]>=0);
	}
}

PAS_OPTIMISER()
void normer_moins_un_un(float * arr, uint n) {
	normer(arr, n);
	FOR(0, i, n) arr[i] = 2*arr[i] - 1;
};

PAS_OPTIMISER()
void prete(float * arr, uint n) {
	float s = rnd()-.5;
	float d = rnd()-.5;
	FOR(0, i, n) {
		if (i%1==0) s += rnd()-.5;
		if (i%2==0) d += rnd()-.5;
		arr[i] = s/2 + d/2;
	}
	normer(arr, n);
};

PAS_OPTIMISER()
void lisser(float * arr, uint n, float A) {
	FOR(0, i, n) {
		arr[i] = A*roundf(arr[i]/A);
	}
};

/*void __5050(float * arr, uint n) {
	assert(n == 49);
};*/

PAS_OPTIMISER()
float lire_flotant(char * fichier) {
	FILE * fp = fopen(fichier, "rb");
	SI_EXISTE(fp, fichier);
	//
	int fd = fileno(fp);
	flock(fd, LOCK_EX);
	//
	float res;
	(void)!fread(&res, sizeof(float), 1, fp);
	//
	flock(fd, LOCK_UN);
	fclose(fp);
	return res;
}

PAS_OPTIMISER()
void ecrire_flotant(char * fichier, float a) {
	FILE * fp = fopen(fichier, "wb");
	//
	int fd = fileno(fp);
	flock(fd, LOCK_EX);
	//
	(void)!fwrite(&a, sizeof(float), 1, fp);
	flock(fd, LOCK_UN);
	fclose(fp);
};

PAS_OPTIMISER()
void ecrire_uint(char * fichier, uint a) {
	FILE * fp = fopen(fichier, "wb");
	//
	int fd = fileno(fp);
	flock(fd, LOCK_EX);
	//
	(void)!fwrite(&a, sizeof(uint), 1, fp);
	flock(fd, LOCK_UN);
	fclose(fp);
};

PAS_OPTIMISER()
uint lire_uint(char * fichier) {
	FILE * fp = fopen(fichier, "rb");
	SI_EXISTE(fp, fichier);
	//
	int fd = fileno(fp);
	flock(fd, LOCK_EX);
	//
	uint res;
	(void)!fread(&res, sizeof(uint), 1, fp);
	flock(fd, LOCK_UN);
	fclose(fp);
	return res;
};

PAS_OPTIMISER()
void lire_N_uint(char * fichier, uint * _uint, uint _N) {
	FILE * fp = fopen(fichier, "rb");
	SI_EXISTE(fp, fichier);
	//
	int fd = fileno(fp);
	flock(fd, LOCK_EX);
	//
	(void)!fread(_uint, sizeof(uint), _N, fp);
	flock(fd, LOCK_UN);
	fclose(fp);
};

PAS_OPTIMISER()
void ecrire_N_uint(char * fichier, uint * _uint, uint _N) {
	FILE * fp = fopen(fichier, "wb");
	//
	int fd = fileno(fp);
	flock(fd, LOCK_EX);
	//
	(void)!fwrite(_uint, sizeof(uint), _N, fp);
	flock(fd, LOCK_UN);
	fclose(fp);
};

//	-- char
PAS_OPTIMISER()
void ecrire_char(char * fichier, char a) {
	FILE * fp = fopen(fichier, "wb");
	//
	int fd = fileno(fp);
	flock(fd, LOCK_EX);
	//
	(void)!fwrite(&a, sizeof(char), 1, fp);
	flock(fd, LOCK_UN);
	fclose(fp);
};

PAS_OPTIMISER()
char lire_char(char * fichier) {
	FILE * fp = fopen(fichier, "rb");
	SI_EXISTE(fp, fichier);
	//
	int fd = fileno(fp);
	flock(fd, LOCK_EX);
	//
	char res;
	(void)!fread(&res, sizeof(char), 1, fp);
	flock(fd, LOCK_UN);
	fclose(fp);
	return res;
};

//	=== Fonctions Variadiques ===

char **   va_allouer_chars(uint N, ...) {
	va_list ptr;
    va_start(ptr, N);

	char ** ret = (char**)malloc(sizeof(char*) * N);
	FOR(0, i, N) {
		ret[i] = va_arg(ptr, char*);
	}

	va_end(ptr);
	return ret;
};

float ** va_allouer_floats(uint N, ...) {
	va_list ptr;
    va_start(ptr, N);

	float ** ret = (float**)malloc(sizeof(float*) * N);
	FOR(0, i, N) {
		ret[i] = va_arg(ptr, float*);
	}

	va_end(ptr);
	return ret;
};

void va_chars_free(uint N, char  **    chars) {
	FOR(0, i, N) free(chars[i]);
	free(chars);
};

void va_uints_free(uint N, float ** flotants) {
	FOR(0, i, N) free(flotants[i]);
	free(flotants);
};

//	=== Matplotlib ===

#define  MATPLOTLIB_COURBES 0
#define MATPLOTLIB_MATRICES 1

static char* demande_tmpt_matplotlib() {
	char * nom_fichier = (char*)malloc(sizeof(char) * 50);
	uint nb = 0;
	FOR(0, x, 100) {
		snprintf(nom_fichier, 50, "tmpt/tempt%i", x);
		FILE * fp = fopen(nom_fichier, "rb");
		if (fp == 0) {
			nb = x + 1;
			break;
		} else {
			fclose(fp);
		}
	}
	if (nb == 0) ERR("Plus de 100 fichier ont ete cree");
	snprintf(nom_fichier, 50, "tmpt/tempt%i", nb-1);
	return nom_fichier;
};

static void lancer_matplotlib(char * fichier, uint mode) {
/*
	Modes :
		0| Courbes avec noms
		1| Matrices avec noms
*/
	char commande[100];
	snprintf(commande, 100, "python3 python_matplotlib.py %s %i", fichier, mode);
	printf("python_matplotlib.py sortie avec %i\n",
		system(commande)
	);
};

void matplotlib_matrices(float ** mat, char ** noms, uint N, uint X, uint Y) {
	char * fichier = demande_tmpt_matplotlib();
	//
	FILE * fp = fopen(fichier, "wb");
	FOPEN_LOCK(fp, fichier);
	//
	FWRITE(&N, sizeof(uint), 1, fp);
	FWRITE(&X, sizeof(uint), 1, fp);
	FWRITE(&Y, sizeof(uint), 1, fp);
	FOR(0, i, N) FWRITE(mat[i], sizeof(float), X * Y, fp);
	FOR(0, i, N) {
		uint _strlen = strlen(noms[i]);
		FWRITE(&_strlen, sizeof(uint), 1,       fp);
		FWRITE(noms[i],   sizeof(char), _strlen, fp);
	}
	//
	FCLOSE_UNCLOCK(fp);
	//
	lancer_matplotlib(fichier, MATPLOTLIB_MATRICES);
	free(fichier);
};

void  matplotlib_courbes(float ** crb, char ** noms, uint L, uint N) {
	char * fichier = demande_tmpt_matplotlib();
	//
	FILE * fp = fopen(fichier, "wb");
	FOPEN_LOCK(fp, fichier);
	//
	FWRITE(&N, sizeof(uint), 1, fp);
	FWRITE(&L, sizeof(uint), 1, fp);
	FOR(0, i, N) FWRITE(crb[i], sizeof(float), L, fp);
	FOR(0, i, N) {
		uint _strlen = strlen(noms[i]);
		FWRITE(&_strlen, sizeof(uint), 1,       fp);
		FWRITE(noms[i],   sizeof(char), _strlen, fp);
	}
	//
	FCLOSE_UNCLOCK(fp);
	//
	lancer_matplotlib(fichier, MATPLOTLIB_COURBES);
	free(fichier);
};