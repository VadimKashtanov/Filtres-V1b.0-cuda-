#include "mdl.cuh"

float          prixs[           PRIXS] = {};
float            ema[EMA_INTS * PRIXS] = {};
float     normalisee[EMA_INTS * PRIXS * N_FLTR] = {};
float dif_normalisee[EMA_INTS * PRIXS * N_FLTR] = {};

ema_int ema_ints[EMA_INTS] = {
	{ 0,    1,    1},
	{ 1,    2,    2},
	{ 2,    4,    4},
	{ 3,    6,    6},
	{ 4,   10,   10},
	{ 5,   20,   20},
	{ 6,   50,   50},
	{ 7,  100,  100},
	{ 8,  200,  200},
	{ 9,  500,  500},
    {10, 1000, 1000}
};

void charger_les_prixs() {
	//	extern float      prixs[         PRIXS         ];
	FILE * fp = fopen("prixs/prixs.bin", "rb");
	ASSERT(fp != 0);
	uint __PRIXS;
	(void)!fread(&__PRIXS, sizeof(uint), 1, fp);
	ASSERT(__PRIXS == PRIXS);
	(void)!fread(prixs, sizeof(float), PRIXS, fp);
	fclose(fp);
};

void calculer_ema_norm_diff() {
	//	extern float        ema[EMA_INTS][    PRIXS    ];
	float k[EMA_INTS];
	float _k[EMA_INTS];
	for (uint i=0; i < EMA_INTS; i++) {
		k[i] = 1.0/(1.0 + (float)ema_ints[i].ema);
		_k[i] = 1.0 - k[i];
		ema[i*PRIXS+0] = prixs[0];

		assert(ema_ints[i].interv <= MAX_INTERVALLE);
	}
	//
	for (uint i=1; i < PRIXS; i++) {
		for (uint j=0; j < EMA_INTS; j++) {
			ema[j*PRIXS+i] = prixs[i]*k[j] + ema[j*PRIXS + i-1]*_k[j];
		};
	};


	//	extern float normalisee[EMA_INTS][PRIXS][N_FLTR];
	float _max, _min;
	FOR(DEPART, t, FIN) {
		FOR(0, e, EMA_INTS) {
			_max = ema[e*PRIXS + t-0*ema_ints[e].interv];
			_min = ema[e*PRIXS + t-0*ema_ints[e].interv];
			FOR(1, i, N_FLTR) {
				if (_max < ema[e*PRIXS + t-i*ema_ints[e].interv])
					_max = ema[e*PRIXS + t-i*ema_ints[e].interv];
				if (_min > ema[e*PRIXS + t-i*ema_ints[e].interv])
					_min = ema[e*PRIXS + t-i*ema_ints[e].interv];
			}
			FOR(0, i, N_FLTR) {
				normalisee[e*PRIXS*N_FLTR+t*N_FLTR+i] = (ema[e*PRIXS+t-i*ema_ints[e].interv]-_min)/(_max-_min);
			}
		};
	};

	FOR(DEPART, t, FIN) {
		FOR(0, e, EMA_INTS) {
			FOR(0, i, N_FLTR-1)
				dif_normalisee[e*PRIXS*N_FLTR+t*N_FLTR+i] = normalisee[e*PRIXS*N_FLTR+t*N_FLTR+i+1]-normalisee[e*PRIXS*N_FLTR+t*N_FLTR+i];
			dif_normalisee[e*PRIXS*N_FLTR+t*N_FLTR+N_FLTR-1] = 0.f;
		}
	}
};

float *          prixs__d = 0x0;
float *            ema__d = 0x0;
float *     normalisee__d = 0x0;
float * dif_normalisee__d = 0x0;

void charger_vram_nvidia() {
	CONTROLE_CUDA(hipMalloc((void**)&prixs__d, sizeof(float) * PRIXS));
	CONTROLE_CUDA(hipMalloc((void**)&ema__d, sizeof(float) * EMA_INTS * PRIXS));
	CONTROLE_CUDA(hipMalloc((void**)&normalisee__d, sizeof(float) * EMA_INTS * PRIXS * N_FLTR));
	CONTROLE_CUDA(hipMalloc((void**)&dif_normalisee__d, sizeof(float) * EMA_INTS * PRIXS * N_FLTR));
	//
	CONTROLE_CUDA(hipMemcpy(         prixs__d,          prixs, sizeof(float) * PRIXS,                     hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(           ema__d,            ema, sizeof(float) * EMA_INTS * PRIXS,          hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(    normalisee__d,     normalisee, sizeof(float) * EMA_INTS * PRIXS * N_FLTR, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(dif_normalisee__d, dif_normalisee, sizeof(float) * EMA_INTS * PRIXS * N_FLTR, hipMemcpyHostToDevice));
};

void     liberer_cudamalloc() {
	CONTROLE_CUDA(hipFree(prixs__d));
	CONTROLE_CUDA(hipFree(ema__d));
	CONTROLE_CUDA(hipFree(normalisee__d));
	CONTROLE_CUDA(hipFree(dif_normalisee__d));
};